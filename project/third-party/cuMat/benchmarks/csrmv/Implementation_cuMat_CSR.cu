#include "benchmark.h"

#include <Eigen/Sparse>
#include <cuMat/Core>
#include <cuMat/Sparse>
#include <iostream>
#include <cstdlib>

void benchmark_cuMat_CSR(
    const std::vector<std::string>& parameterNames,
    const Json::Array& parameters,
    const std::vector<std::string>& returnNames,
    Json::Array& returnValues)
{
    //number of runs for time measures
    const int runs = 10;
	const int subruns = 10;

    int numConfigs = parameters.Size();
    for (int config = 0; config < numConfigs; ++config)
    {
		//Input
		int gridSize = parameters[config][0].AsInt32();
		double totalTime = 0;
		std::cout << "  Grid Size: " << gridSize << std::flush;
		int matrixSize = gridSize * gridSize;

		//Create matrix
#define IDX(x, y) ((y) + (x)*gridSize)
		Eigen::SparseMatrix<float, Eigen::RowMajor, int> matrix(matrixSize, matrixSize);
		matrix.reserve(Eigen::VectorXi::Constant(matrixSize, 5));
		for (int x = 0; x<gridSize; ++x) for (int y = 0; y<gridSize; ++y)
		{
			int row = IDX(x, y);
			if (x > 0) matrix.insert(row, IDX(x - 1, y)) = -1;
			if (y > 0) matrix.insert(row, IDX(x, y - 1)) = -1;
			matrix.insert(row, row) = 4;
			if (y < gridSize - 1) matrix.insert(row, IDX(x, y + 1)) = -1;
			if (x < gridSize - 1) matrix.insert(row, IDX(x + 1, y)) = -1;
		}
		matrix.makeCompressed();

		//Create vector
		Eigen::VectorXf ex = Eigen::VectorXf::Random(matrixSize);

		//Send to cuMat
		typedef cuMat::SparseMatrix<float, 1, cuMat::CSR> SMatrix;
		typedef cuMat::SparsityPattern<cuMat::CSR> SPattern;
		SPattern pattern;
		pattern.rows = matrixSize;
		pattern.cols = matrixSize;
		pattern.nnz = matrix.nonZeros();
		pattern.JA = SPattern::IndexVector(matrixSize + 1); pattern.JA.copyFromHost(matrix.outerIndexPtr());
		pattern.IA = SPattern::IndexVector(pattern.nnz); pattern.IA.copyFromHost(matrix.innerIndexPtr());
        pattern.assertValid();
		SMatrix mat(pattern);
		mat.getData().copyFromHost(matrix.valuePtr());

		cuMat::VectorXf x = cuMat::VectorXf::fromEigen(ex);
		cuMat::VectorXf r(matrixSize);

        //Run it multiple times
        for (int run = 0; run < runs; ++run)
        {
            //Main logic
			hipDeviceSynchronize();
			auto start = std::chrono::steady_clock::now();

			for (int i = 0; i < subruns; ++i) {
				r.inplace() = mat * x;
			}

			hipDeviceSynchronize();
			auto finish = std::chrono::steady_clock::now();
			double elapsed = std::chrono::duration_cast<
				std::chrono::duration<double>>(finish - start).count() * 1000 / subruns;

            totalTime += elapsed;
        }

        //Result
        Json::Array result;
        double finalTime = totalTime / runs;
        result.PushBack(finalTime);
        returnValues.PushBack(result);
        std::cout << " -> " << finalTime << "ms" << std::endl;
    }
}