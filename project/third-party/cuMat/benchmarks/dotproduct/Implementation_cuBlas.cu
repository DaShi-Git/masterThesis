#include "benchmark.h"

#include <cuMat/Core>
#include <iostream>
#include <cstdlib>
#include <hipblas.h>

namespace {

//copied from cuMat/src/CublasApi.h

static const char* getErrorName(hipblasStatus_t status)
{
    switch (status)
    {
    case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED: cuBLAS was not initialized";
    case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED: resource allocation failed";
    case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE: invalid value was passed as argument";
    case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH: device architecture not supported";
    case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR: access to GPU memory failed";
    case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED: general kernel launch failure";
    case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR: an internal error occured";
    case HIPBLAS_STATUS_NOT_SUPPORTED: return "HIPBLAS_STATUS_NOT_SUPPORTED: functionality is not supported";
    case HIPBLAS_STATUS_UNKNOWN: return "HIPBLAS_STATUS_UNKNOWN: required licence was not found";
    default: return "";
    }
}
static void cublasSafeCall(hipblasStatus_t status, const char *file, const int line)
{
    if (HIPBLAS_STATUS_SUCCESS != status) {
        std::string msg = cuMat::internal::ErrorHelpers::format("cublasSafeCall() failed at %s:%i : %s\n",
            file, line, getErrorName(status));
        std::cerr << msg << std::endl;
        throw cuMat::cuda_error(msg);
    }
}
#define CUBLAS_SAFE_CALL( err ) cublasSafeCall( err, __FILE__, __LINE__ )

}

//Benchmark with cuBLAS
//cuMat is used to allocate the matrices, but the computation is done in cuBLAS (axpy)
void benchmark_cuBlas(
    const std::vector<std::string>& parameterNames,
    const Json::Array& parameters,
    const std::vector<std::string>& returnNames,
    Json::Array& returnValues)
{
    //number of runs for time measures
    const int runs = 10;
	const int subruns = 10;

    //test if the config is valid
    assert(parameterNames.size() == 1);
    assert(parameterNames[0] == "Vector-Size");
    assert(returnNames.size() == 1);
    assert(returnNames[0] == "Time");

    cuMat::SimpleRandom rand;

    int numConfigs = parameters.Size();
    for (int config = 0; config < numConfigs; ++config)
    {
		//Input
		int vectorSize = parameters[config][0].AsInt32();
		double totalTime = 0;
		std::cout << "  VectorSize: " << vectorSize << std::flush;

		//Create matrices
		cuMat::VectorXf a(vectorSize); rand.fillUniform(a, 0, 1);
		cuMat::VectorXf b(vectorSize); rand.fillUniform(b, 0, 1);
		float* aRaw = a.data();
		float* bRaw = b.data();
        
        //create cuBLAS handle
        hipblasHandle_t handle = nullptr;
        CUBLAS_SAFE_CALL(hipblasCreate(&handle));
		CUBLAS_SAFE_CALL(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

        //Run it multiple times
        for (int run = 0; run < runs; ++run)
        {

            //Main logic
			hipDeviceSynchronize();
			auto start = std::chrono::steady_clock::now();
            
            //pure cuBLAS + CUDA:
			for (int subrun = 0; subrun < subruns; ++subrun) {
				float result;
				CUBLAS_SAFE_CALL(hipblasSdot(handle, vectorSize, aRaw, 1, bRaw, 1, &result));
			}

            hipDeviceSynchronize();
			auto finish = std::chrono::steady_clock::now();
			double elapsed = std::chrono::duration_cast<
				std::chrono::duration<double>>(finish - start).count() * 1000 / subruns;

            totalTime += elapsed;
        }
        
        CUBLAS_SAFE_CALL(hipblasDestroy(handle));

        //Result
        Json::Array result;
        double finalTime = totalTime / runs;
        result.PushBack(finalTime);
        returnValues.PushBack(result);
        std::cout << " -> " << finalTime << "ms" << std::endl;
    }
}
