#include "hip/hip_runtime.h"
#include <catch/catch.hpp>
#include <limits>

#include <cuMat/Core>

#include "Utils.h"

using namespace cuMat;

TEST_CASE("random", "[random]")
{
    SimpleRandom r;

    SECTION("bool") {
        BMatrixXb m(100, 110, 120);
        r.fillUniform(m, false, true);
        //there is not much we can test here
    }

    SECTION("int")
    {
        BMatrixXi m(100, 110, 120);
        r.fillUniform(m, -10, 50);
        REQUIRE(-10 <= (int)m.minCoeff());
        REQUIRE(50 > (int)m.maxCoeff());
    }

    SECTION("long long")
    {
        BMatrixXll m(100, 110, 120);
        r.fillUniform(m, -100, 500);
        REQUIRE(-100 <= (long long)m.minCoeff());
        REQUIRE(500 > (long long)m.maxCoeff());
    }

    SECTION("float")
    {
        BMatrixXf m(100, 110, 120);
        r.fillUniform(m, 5.5, 12.5);
        REQUIRE(5.5 - 0.00001 <= (float)m.minCoeff());
        REQUIRE(12.5 + 0.00001 > (float)m.maxCoeff());
    }

    SECTION("double")
    {
        BMatrixXd m(100, 110, 120);
        r.fillUniform(m, -5.5, 12.5);
        REQUIRE(-5.5 - 0.00001 <= (double)m.minCoeff());
        REQUIRE(12.5 + 0.00001 > (double)m.maxCoeff());
    }

    SECTION("complex-float")
    {
        BMatrixXcf m(100, 110, 120);
        r.fillUniform(m, cfloat(5.5, 10.5), cfloat(12.5, 22.5));
        REQUIRE(5.5 - 0.00001 <= (float)m.real().minCoeff());
        REQUIRE(12.5 + 0.00001 > (float)m.real().maxCoeff());
        REQUIRE(10.5 - 0.00001 <= (float)m.imag().minCoeff());
        REQUIRE(22.5 + 0.00001 > (float)m.imag().maxCoeff());
    }

    SECTION("complex-double")
    {
        BMatrixXcd m(100, 110, 120);
        r.fillUniform(m, cdouble(5.5, 10.5), cdouble(12.5, 22.5));
        REQUIRE(5.5 - 0.00001 <= (double)m.real().minCoeff());
        REQUIRE(12.5 + 0.00001 > (double)m.real().maxCoeff());
        REQUIRE(10.5 - 0.00001 <= (double)m.imag().minCoeff());
        REQUIRE(22.5 + 0.00001 > (double)m.imag().maxCoeff());
    }
}

TEST_CASE("random-defaults", "[random]")
{
    SimpleRandom r;

    SECTION("bool") {
        BMatrixXb m(100, 110, 120);
        r.fillUniform(m);
        //there is not much we can test here
    }

    SECTION("int")
    {
        BMatrixXi m(100, 110, 120);
        r.fillUniform(m);
        REQUIRE(0 <= (int)m.minCoeff());
        REQUIRE(std::numeric_limits<int>::max() > (int)m.maxCoeff());
    }

    SECTION("float")
    {
        BMatrixXf m(100, 110, 120);
        r.fillUniform(m);
        REQUIRE(0 - 0.00001 <= (float)m.minCoeff());
        REQUIRE(1 + 0.00001 > (float)m.maxCoeff());
    }

    SECTION("double")
    {
        BMatrixXd m(100, 110, 120);
        r.fillUniform(m);
        REQUIRE(0 - 0.00001 <= (double)m.minCoeff());
        REQUIRE(1 + 0.00001 > (double)m.maxCoeff());
    }

    SECTION("complex-float")
    {
        BMatrixXcf m(100, 110, 120);
        r.fillUniform(m);
        REQUIRE(0 - 0.00001 <= (float)m.real().minCoeff());
        REQUIRE(1 + 0.00001 > (float)m.real().maxCoeff());
        REQUIRE(0 - 0.00001 <= (float)m.imag().minCoeff());
        REQUIRE(1 + 0.00001 > (float)m.imag().maxCoeff());
    }

    SECTION("complex-double")
    {
        BMatrixXcd m(100, 110, 120);
        r.fillUniform(m);
        REQUIRE(0 - 0.00001 <= (double)m.real().minCoeff());
        REQUIRE(1 + 0.00001 > (double)m.real().maxCoeff());
        REQUIRE(0 - 0.00001 <= (double)m.imag().minCoeff());
        REQUIRE(1 + 0.00001 > (double)m.imag().maxCoeff());
    }
}