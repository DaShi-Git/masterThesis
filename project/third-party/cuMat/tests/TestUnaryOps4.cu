#include "hip/hip_runtime.h"
#include <catch/catch.hpp>
#include <vector>
//#include <unsupported/Eigen/SpecialFunctions>

#include <cuMat/Core>

#include "Utils.h"
#include "TestUnaryOps.cuh"

//UNARY_TEST_CASE_FLOAT(cwiseErf, erf, -1000.0, 1000); //throws THIS_TYPE_IS_NOT_SUPPORTED
//UNARY_TEST_CASE_FLOAT(cwiseErfc, erfc, -1000.0, 1000);
//UNARY_TEST_CASE_FLOAT(cwiseLgamma, lgamma, 0, 10);
//more special functions are not supported directly in CUDA
//Except the bessel functions are supported, but not supported in Eigen (no way to test them easily)
//TODO: add more special functions
