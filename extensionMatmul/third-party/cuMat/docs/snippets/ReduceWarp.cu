#include "hip/hip_runtime.h"
template<typename _Input, typename _Output, typename _Op, typename _Scalar, int _Axis>
__global__ void ReduceWarpKernel(
	_Input input, _Output output, _Op op,
	Index I_, int N, Index S)
{
	CUMAT_KERNEL_1D_LOOP(i_, I_)
		const Index i = i_ / 32;
		const int warp = i_ % 32;
		const Index O = Offset(i);
		//local reduce
		_Scalar v = initial;
		for (int n = warp; n < N; n += 32)
			v = op(v, input[n*S + O]);
		//final warp reduce
		#pragma unroll
		for (int offset = 16; offset > 0; offset /= 2)
			v += __shfl_down_sync(0xffffffff, v, offset);
		//write output
		if (warp == 0) output[i] = v;
	CUMAT_KERNEL_1D_LOOP_END
}
//Warp size is always 32 in all Nvidia cards.
//Launched with optimal threads-per-block and grid size for I_=I*32 threads, 
//as determined by hipOccupancyMaxPotentialBlockSize